#include <hip/hip_runtime.h>

#include <iostream>
#include <random>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, char const *func, char const *file, int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(char const *file, int line) {
  hipError_t const err{hipGetLastError()};
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

template <int threadsPerBlock>
__global__ void vectorCopyVectorized(float4 *input, float4 *output, int N) {
  const int i = threadIdx.x + blockIdx.x * threadsPerBlock;

  if (i < (N >> 2)) {
    output[i] = input[i];
  }
}

template <int threadsPerBlock>
void launchVectorCopyVectorized(float *input, float *output, int N) {
  const int blocksPerGrid = (N / 4 + threadsPerBlock - 1) / threadsPerBlock;
  vectorCopyVectorized<threadsPerBlock><<<blocksPerGrid, threadsPerBlock>>>(
      reinterpret_cast<float4 *>(input), reinterpret_cast<float4 *>(output), N);
}

bool checkCorrectness(float *input, float *output, int N) {
  for (int i = 0; i < N; i++) {
    if (fabs(input[i] - output[i]) > 1e-5) {
      std::cout << "Verification failed" << std::endl;
      return false;
    }
  }
  std::cout << "Verification passed" << std::endl;
  return true;
}

int main() {
  const int N = 1 << 30;
  const size_t size = N * sizeof(float);
  const int threadsPerBlock = 1 << 10;

  float *inputHost = new float[N];
  float *outputHost = new float[N];

  std::default_random_engine generator(42);
  std::normal_distribution<float> distribution(0.0, 1.0);

  for (int i = 0; i < N; i++) {
    inputHost[i] = distribution(generator);
  }

  float *inputDevice;
  float *outputDevice;

  CHECK_CUDA_ERROR(hipMalloc(&inputDevice, size));
  CHECK_CUDA_ERROR(hipMalloc(&outputDevice, size));

  CHECK_CUDA_ERROR(
      hipMemcpy(inputDevice, inputHost, size, hipMemcpyHostToDevice));

  launchVectorCopyVectorized<threadsPerBlock>(inputDevice, outputDevice, N);

  CHECK_LAST_CUDA_ERROR();

  CHECK_CUDA_ERROR(
      hipMemcpy(outputHost, outputDevice, size, hipMemcpyDeviceToHost));

  if (!checkCorrectness(inputHost, outputHost, N)) {
    return -1;
  }

  hipEvent_t start, stop;
  int numWarmup = 1000;
  int numRounds = 10000;
  size_t numCrossMemoryBounds = 2 * size;
  float time;

  for (int i = 0; i < numWarmup; i++) {
    launchVectorCopyVectorized<threadsPerBlock>(inputDevice, outputDevice, N);
  }

  CHECK_LAST_CUDA_ERROR();

  CHECK_CUDA_ERROR(hipEventCreate(&start));
  CHECK_CUDA_ERROR(hipEventCreate(&stop));

  CHECK_CUDA_ERROR(hipEventRecord(start));
  for (int i = 0; i < numRounds; i++) {
    launchVectorCopyVectorized<threadsPerBlock>(inputDevice, outputDevice, N);
  }
  CHECK_CUDA_ERROR(hipEventRecord(stop));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop));
  CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));

  float latency = time / numRounds;
  float bandwidth = (numCrossMemoryBounds / latency) / 1e6;

  std::cout << "Latency = " << latency << " ms" << std::endl;
  std::cout << "Bandwidth = " << bandwidth << " GB/s" << std::endl;
  std::cout << "% of max = " << bandwidth / 3300 * 100 << " %" << std::endl;

  CHECK_CUDA_ERROR(hipFree(inputDevice));
  CHECK_CUDA_ERROR(hipFree(outputDevice));

  free(inputHost);
  free(outputHost);

  return 0;
}